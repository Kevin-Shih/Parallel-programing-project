#include "hip/hip_runtime.h"
#include "kernel_util.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace kernel_utils {

double distance(Position const &pos_1, Position const &pos_2)
{
    Position pos_diff = pos_1 - pos_2;

    return sqrt(pow(pos_diff.x, 2) + pow(pos_diff.y, 2));
}

std::vector<float> get_bound(Position point, double radius)
{
    std::vector<float> bounds(4);
    bounds[0] = point.x - radius;
    bounds[1] = point.y - radius;
    bounds[2] = point.x + radius;
    bounds[3] = point.y + radius;
    return bounds;
}

}

Tree::~Tree()
{
}

TreeNode *get_new_node(TreeNode *start, TreeNode *target, double step_size, double min_step_size)
{
    Position pos_diff = target->pos - start->pos;
    double dist = kernel_utils::distance(start->pos, target->pos);
    if (dist < min_step_size) {
        return nullptr;
    }
    double length = min(step_size, dist);
    Position vec_step = (start->pos + pos_diff * (length / dist));
    TreeNode *new_node = new TreeNode(vec_step);
    new_node->parent = start;
    start->child.push_back(new_node);
    return new_node;
}

TreeNode * random_position(Position const &target, float std, std::mt19937 &generator)
{

    Position tmp_pos = {-1, -1};
    while (tmp_pos.x >= 1500 || tmp_pos.x < 0) {
        tmp_pos.x = kernel_utils::normal(target.x, std, generator);
    }
    while (tmp_pos.y >= 1000 || tmp_pos.y < 0) {
        tmp_pos.y = kernel_utils::normal(target.y, std, generator);
    }
    TreeNode *new_node = new TreeNode(tmp_pos);
    return new_node;
}

bool node_near_obstacle(int **map, const TreeNode *node, float radius) {
    return point_near_obstacle(map, node->pos, radius);
}

__global__ void near_obstacle(int *obs_flag, int **map, int low_x, int low_y) {
    const int thisX = low_x + threadIdx.x + blockDim.x * blockIdx.x;
    const int thisY = low_y + threadIdx.y + blockDim.y * blockIdx.y;
    __shared__ uint8_t flag;
    flag = 0;
    if (map[thisY][thisX] == 0)
        flag = 1;
    __syncthreads();
    if(thisX == low_x && thisY == low_y){
        *obs_flag = flag;
    }
}

bool point_near_obstacle(int **map, const Position& point, float radius) {
    auto vec = kernel_utils::get_bound(point, radius);

    float low_x = max(0, (int)rint(vec[0]));
    float low_y = max(0, (int)rint(vec[1]));
    float high_x = min((int)1500 - 1, (int)rint(vec[2]));
    float high_y = min((int)1000 - 1, (int)rint(vec[3]));
    int flag = 0;
    dim3 threadsPerBlock(16, 16);
    dim3 numsBlock((high_x-low_x+threadsPerBlock.x-1) / threadsPerBlock.x, (high_y-low_y+threadsPerBlock.y-1) / threadsPerBlock.y);
    int **d_map, *d_flag;
    hipHostRegister(map, 1500*1000*sizeof(int), hipHostRegisterMapped);
    hipHostGetDevicePointer(&d_map, map, 0);
    hipHostRegister(&flag, sizeof(int), hipHostRegisterMapped);
    hipHostGetDevicePointer(&d_flag, &flag, 0);
    near_obstacle<<<numsBlock, threadsPerBlock>>>(d_flag, d_map, low_x, low_y);

    return flag;
}

bool intersection(int **map, const Position& start, const Position& end, float radius) {
    double dist = kernel_utils::distance(start, end);
    int num_points = static_cast<int>(dist);
    for (int i = 0; i <= num_points; ++i) {
        Position point = {
            start.x + static_cast<int>((end.x - start.x) * i / num_points),
            start.y + static_cast<int>((end.y - start.y) * i / num_points)
        };

        if (point_near_obstacle(map, point, radius)) {
            return true;
        }
    }
    return false;
}

TreeNode* nearest(TreeNode* root, const TreeNode* target, int **map, float radius) {
    TreeNode* nearest_node = nullptr;
    double min_dist = std::numeric_limits<double>::infinity();

    std::queue<TreeNode*> queue;
    queue.push(root);

    while (!queue.empty()) {
        TreeNode* current = queue.front();
        queue.pop();

        if (!intersection(map, current->pos, target->pos, radius)) {
            double dist = kernel_utils::distance(current->pos, target->pos);
            if (dist < min_dist) {
                min_dist = dist;
                nearest_node = current;
            }
        }

        for (TreeNode* child : current->child) {
            queue.push(child);
        }
    }

    return nearest_node;
}

// void grow_tree(TreeNode **node_list, int **map, Tree *tree, 
//             float radius, float step_size, float std, std::mt19937 &generator){
//     int thisX = blockIdx.x * blockDim.x + threadIdx.x;
//     TreeNode *root = tree->root, *end = tree->end;
//     TreeNode *near_node = NULL, *new_node = NULL;
//     TreeNode *rand_node = random_position(end->pos, std, generator);
//     if (!node_near_obstacle(map, rand_node, radius)) { 
//         // if is valid rand_node
//         near_node = nearest(root, rand_node, map, radius);
//         if (near_node) { 
//             // if has valid near_node
//             new_node = get_new_node(near_node, rand_node, step_size);
//         }
//     }
//     node_list[thisX] = new_node;
// }

// void check_valid(TreeNode *near_node, const std::vector<std::vector<int>>& map, Tree *tree, 
//             float radius, float step_size, float std, std::mt19937 &generator){
//     int thisX = blockIdx.x * blockDim.x + threadIdx.x;
//     if (!node_near_obstacle(map, rand_node, radius)) { 
//         // if is valid rand_node
//         near_node = nearest(root, rand_node, map, radius);
        
//     }
// }